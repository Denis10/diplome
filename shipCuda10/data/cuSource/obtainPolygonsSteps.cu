
#include <hip/hip_runtime.h>
extern "C"
#define n (3)
#define qPoints (58)
#define qPolygons (96)
__global__ void obtainPolygonsSteps(int* dev_S,int* dev_polygonToFillX,int* dev_polygonToFillY,int* dev_shipLocationX,
int* dev_shipLocationZ,float* dev_matrixC,int* dev_points,int* dev_polygons,int* dev_normals,const int N)
{	
	int j = threadIdx.x;
	if (j<N){		
		//__shared__ 
		float observerCoord[3*qPoints];
		for (int i = 0; i < qPoints; i++) {
			observerCoord[3*i] = (dev_points[3*i]+dev_shipLocationX[j]) * dev_matrixC[0] + dev_points[3*i+1]
			* dev_matrixC[1] + (dev_points[3*i+2]+dev_shipLocationZ[j]) * dev_matrixC[2] +dev_matrixC[3];			
			observerCoord[3*i+1] = (dev_points[3*i]+dev_shipLocationX[j]) * dev_matrixC[4] + dev_points[3*i+1]
			* dev_matrixC[5] + (dev_points[3*i+2]+dev_shipLocationZ[j]) * dev_matrixC[6] + dev_matrixC[7];
			observerCoord[3*i+2] = (dev_points[3*i]+dev_shipLocationX[j]) * dev_matrixC[8] + dev_points[3*i+1]
			* dev_matrixC[9] + (dev_points[3*i+2]+dev_shipLocationZ[j]) * dev_matrixC[10] + dev_matrixC[11];
		}
		/*
		float to2d[2*qPoints];
		for (int i = 0; i < qPoints; i++) {
			to2d[2*i] = 0.6 * observerCoord[3*i+2] / observerCoord[3*i];
			to2d[2*i+1] = 0.6 * observerCoord[3*i+1] / observerCoord[3*i];
		}
		// ---------------to2dmm----------------------------------------------
		double to2dmm[2*qPoints];
		for (int i = 0; i < qPoints; i++) {
			to2dmm[2*i] = 160 * to2d[2*i];
			to2dmm[2*i+1] = 160 * to2d[2*i+1];
		}
		// -----------------------spX, spY--------------------------------------
		int spX[qPoints];
		int spY[qPoints];
		for (int i = 0; i < qPoints; i++) {
			spX[i] = (int) (4.35 * to2dmm[2*i]);
			spY[i] = (int)(4.35 * to2dmm[2*i+1]);
		}
		int xp[qPoints];
		int yp[qPoints];
		for (int i = 0; i < qPoints; i++) {
			xp[i] = spX[i] + 350;
			yp[i] = -spY[i] + 350;
		}	*/	
		int xp[qPoints];
		int yp[qPoints];
		for (int i = 0; i < qPoints; i++) {
			xp[i] = (int) (4.35f * 160 * 0.6f * observerCoord[3*i+2] / observerCoord[3*i]) + 350;
			yp[i] = -(int) (4.35f * 160 * 0.6f * observerCoord[3*i+1] / observerCoord[3*i]) + 350;
		}
		float NormalObs[3*qPolygons];
		for (int i = 0; i < qPolygons; i++) {
			NormalObs[3*i] = dev_normals[3*i] * dev_matrixC[0] + dev_normals[3*i+1]
					* dev_matrixC[1] + dev_normals[3*i+2] * dev_matrixC[2];
			NormalObs[3*i+1] = dev_normals[3*i] * dev_matrixC[4] + dev_normals[3*i+1]
					* dev_matrixC[5] + dev_normals[3*i+2] *dev_matrixC[6];
			NormalObs[3*i+2] = dev_normals[3*i] * dev_matrixC[8] + dev_normals[3*i+1]
					* dev_matrixC[9] + dev_normals[3*i+2] * dev_matrixC[10];
		}
		for (int i = 0; i < qPolygons; i++) {
			if (
				(
				NormalObs[3*i] * observerCoord[3*dev_polygons[3*i]] + NormalObs[3*i+1]
					*observerCoord[3*dev_polygons[3*i]+1] + NormalObs[3*i+2]
					* observerCoord[3*dev_polygons[3*i]+2]
				)<0){
					dev_S[j*qPolygons+i]=1;
			}
			else {
			dev_S[j*qPolygons+i]=0;
			}
		}
		for (int i = 0; i < qPolygons; i++) {
			dev_polygonToFillX[j*qPolygons*n+3*i] = xp[dev_polygons[3*i]];
			dev_polygonToFillX[j*qPolygons*n+3*i+1] = xp[dev_polygons[3*i+1]];
			dev_polygonToFillX[j*qPolygons*n+3*i+2] = xp[dev_polygons[3*i+2]];
			dev_polygonToFillY[j*qPolygons*n+3*i] = yp[dev_polygons[3*i]];
			dev_polygonToFillY[j*qPolygons*n+3*i+1] = yp[dev_polygons[3*i+1]];
			dev_polygonToFillY[j*qPolygons*n+3*i+2] = yp[dev_polygons[3*i+2]];
		}			
	}
}