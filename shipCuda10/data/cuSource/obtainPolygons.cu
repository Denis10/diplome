
#include <hip/hip_runtime.h>
extern "C"
#define n (3)
#define qPoints (58)
#define qPolygons (96)
__global__ void obtainPolygons(int* dev_S,int* dev_polygonToFillX,int* dev_polygonToFillY,int* dev_shipLocationX,
int* dev_shipLocationZ,float* dev_matrixC,int* dev_points,int* dev_polygons,int* dev_normals,const int N)
{	
	int j = threadIdx.x;
	if (j<N){		
		//__shared__ 
		float observerCoord[3*qPoints];
		for (int i = 0; i < qPoints; i++) {
			observerCoord[3*i] = (dev_points[3*i]+dev_shipLocationX[j]) * dev_matrixC[0] + dev_points[3*i+1]
			* dev_matrixC[1] + (dev_points[3*i+2]+dev_shipLocationZ[j]) * dev_matrixC[2] +dev_matrixC[3];			
			observerCoord[3*i+1] = (dev_points[3*i]+dev_shipLocationX[j]) * dev_matrixC[4] + dev_points[3*i+1]
			* dev_matrixC[5] + (dev_points[3*i+2]+dev_shipLocationZ[j]) * dev_matrixC[6] + dev_matrixC[7];
			observerCoord[3*i+2] = (dev_points[3*i]+dev_shipLocationX[j]) * dev_matrixC[8] + dev_points[3*i+1]
			* dev_matrixC[9] + (dev_points[3*i+2]+dev_shipLocationZ[j]) * dev_matrixC[10] + dev_matrixC[11];
		}
		int xp[qPoints];
		int yp[qPoints];
		for (int i = 0; i < qPoints; i++) {
			xp[i] = (int) (4.35f * 160 * 0.6f * observerCoord[3*i+2] / observerCoord[3*i]) + 350;
			yp[i] = -(int) (4.35f * 160 * 0.6f * observerCoord[3*i+1] / observerCoord[3*i]) + 350;
		}		
		for (int i = 0; i < qPolygons; i++) {
			if (
				(
				(dev_normals[3*i] * dev_matrixC[0] + dev_normals[3*i+1] * dev_matrixC[1] + dev_normals[3*i+2] * dev_matrixC[2]) * 
				observerCoord[3*dev_polygons[3*i]] + 
				(dev_normals[3*i] * dev_matrixC[4] + dev_normals[3*i+1]* dev_matrixC[5] + dev_normals[3*+2] * dev_matrixC[6]) * 
				observerCoord[3*dev_polygons[3*i]+1] + 
				(dev_normals[3*i] * dev_matrixC[8] + dev_normals[3*i+1]	* dev_matrixC[9] + dev_normals[3*i+2] * dev_matrixC[10]) *
				observerCoord[3*dev_polygons[3*i]+2])<0){
					dev_S[j*qPolygons+i]=1;
			}
			else {
			dev_S[j*qPolygons+i]=0;
			}
		}
		for (int i = 0; i < qPolygons; i++) {
			dev_polygonToFillX[j*qPolygons*n+3*i] = xp[dev_polygons[3*i]];
			dev_polygonToFillX[j*qPolygons*n+3*i+1] = xp[dev_polygons[3*i+1]];
			dev_polygonToFillX[j*qPolygons*n+3*i+2] = xp[dev_polygons[3*i+2]];
			dev_polygonToFillY[j*qPolygons*n+3*i] = yp[dev_polygons[3*i]];
			dev_polygonToFillY[j*qPolygons*n+3*i+1] = yp[dev_polygons[3*i+1]];
			dev_polygonToFillY[j*qPolygons*n+3*i+2] = yp[dev_polygons[3*i+2]];
		}			
	}
}