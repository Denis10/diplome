
#include <hip/hip_runtime.h>
#define size (8000)
extern "C"

__device__ float vxd(const float m,float Vy, float w, float ZpTei, float Xr, float Kvx)
{
	if (Vy > 0) {
            return ((m * Vy * w - Xr + ZpTei) * Kvx);
        } else {
            return ((m * Vy * w * 1.09f - Xr + ZpTei) * Kvx);//Vx*1.061 ;N0=3;k11=580.91f Ubuntu
            //return ((m * Vy * w  - Xr + ZpTei) * Kvx);
        }//*1.1045; k11=5.91f windows ; 1.09; k11=580.91f
}

extern "C"

__device__ float vyd(const float m,float Vx, float w, float ZpYri, float Yr, float Kvy) {
        if (Vx > 0) {
            return ((Yr - ZpYri - m * Vx * w * 1.0165f) * Kvy);//Vx*1.0179
            //return ((Yr - ZpYri - m * Vx * w) * Kvy);
        } else {//*1.0165; k11=5.91f windows
            return ((Yr - ZpYri - m * Vx * w) * Kvy);
        }
    }
extern "C"

__device__ float wd(const float lr,float Mr, float ZpYri, float Yr, float Kw) {
        return ((Mr + lr * ZpYri) * Kw);
    }
	
extern "C"

__global__ void ParallelSpeedCalcCuda(float* VX, float* VY, float* ww, float* X, float* Y,
float* W, float* Xobs, float* Yobs,const int nShips) 
{ 
int j = blockDim.x * blockIdx.x + threadIdx.x; 
if (j<nShips){
		float k11, k22, k66, k26;
        
		float devVX[size];
		float devVY[size];
		float devww[size];
		float devX[size];
		float devY[size];
		float devW[size];
		float devXobs[size];
		float devYobs[size];
		
		const float T = 4.0f;
		const float L = 80.0f;
		const float B = 15.0f;// ������, �����, ������
															// c 323, 321
		const float m = 20000000000.0f; // ����� �����
		const float lr = 40.0f; // ���������� �� ��� �� �� �����
		float Vx = 5.0f, Vy = 0.0f;// �������� �� ����
		float v = 0.0f, w = 0.0f;// �������� � ������� ��������
		
        const float delta = 0.7f;//����������� ����� �������
        float c1, c2, c3, m1, m2;//����������� �.�.�������
        float w_ = 0.0f;//���� ������						approx
        float betta_d = 0.0f;//���� ������ � ������ ������� (rad)
        float Cxr, Cyr, Cmr;//������������ Cxr, ������� ����, �������
        //	float L1=B,T1=L/2;//�����, ��������� ������������� ����� ????????????????????
        const float p = 1000.0f;//��������� ����
        float Xr, Yr, Mr;//����������������� ������
        float ZpTei = 0;//�������� ���� ������ ��� ����������� ������������� �������� =R+Xa
        float Jz;//������ ������� ����� ����� ������������ ��� Gz
        float ZpYri;//������� ���� ��� 
        const float a66 = 0.31f;//�������� �����������
        //float Ramp = 0.0f;//��� ��������� �����
        const float lyamdaR = 1.4f;//��������� c 335 //0.5--1.4 //1
        
        const float deltaR = 0.349f;//���� ���������� ����  - 20% //0.349f
        
//	float Va;//�������� ��������� ���� �� ����
        float Yri, Ysi;//������� � ��������������� ����
        const float Ar = 6.0f;//������� ������//5 ����� ���������
        const float D = 1.5f;//������� ����� �����//2 ����� ���������
        const float Cta = 10.0f, Ctt = 50.0f;//c 334 //Cta<=20, 0.5--30, Ctt<=20, 1--50 UP!!!
        //Ctt =30
        float Kvx, Kvy, Kw;//����������� ��� �����������
        const float No = 3.0f;//������� ��������� //������//3 �������
        const float Re = 5000000.0f;//����� ��������� >5E8
        float K_betta;
        float fit;
        const float xk = 1.0f;
        const float bettar = 0.9f;
        const float fik = 0.95f;
        float ld_;//������������� ����� ������� c228
        float betta_D;//����������� ����������
        float fiD;//c 338
        float CyD1;
        float CyD;
        float A0;//������� ����� �����
        float xD;//c 339
        float viv = 0.0f;//���� ��� ����������� ������� ��������� � 27 ch3_2
		float Rmatr[3][3]={cosf(viv),-sinf(viv),0.0f,sinf(viv),cosf(viv),0.0f,0.0f,0.0f,1.0f};
		 //------------------------------------------------------------
//c 330         5.91-� �����, 580.91 - ��������� �
        k11 = (580.91f * (float) pow(B / L, 2.0f) + 7.76f * (B / L) - 0.259f) / (48.4f - 6.89f * (B / T) + 1.47f * (float) pow(B / T, 2.0f) - 0.0475f * (float) pow(B / T, 3.0f));

        k22 = ((0.722f + 0.224f * delta) * (1.022f - (float) pow(B / L, 2.0f))) / (0.264f + 0.368f * (B / T));
        k66 = (2.0f * T / B) * (2.59f + 0.781f * delta) * (0.357f - 1.77f * (float) pow(B / L, 2.0f));
        k26 = k22;
        //k26=0;       
//c 323
        c1 = 3.14f * (T / L) * (float) pow((0.63f / delta), (5.0f / 2.0f)) * (float) pow(L / (6.0f * B), (1.0f / 3.0f)) - 0.032f;
        c2 = -2.0f * k11 * delta * (B / L);
        c3 = 1.35f * (float) pow(T / B, (1.0f / 2.0f)) * (float) pow((0.63f / delta), (3.0f / 2.0f)) - 0.029f;
        m1 = 1.67f * (T / L) - 0.56f * delta + 0.43f;
        m2 = -0.44f * (T / L) - 0.0375f;
        // System.out.printf("c1=%f\tc2=%f\tc3=%f\tm1=%f\tm2=%f\n", c1, c2, c3, m1, m2);
        Jz = (m * (float) pow(L, 2.0f) / 12.4f) * (0.463f + 0.574f * (float) pow(delta, a66) + (float) pow(B / L, 2.0f));//c 330
        
        Kvx = 1 / (m * (1 + k11));
        Kvy = 1 / (m * (1 + k22));
        Kw = 1 / (Jz * (1 + k66));//????? m        
//----------------------------------------------------------------
        float k1, k2, k3, k4;
        float q1, q2, q3, q4;
        float z1, z2, z3, z4;
        float j1, j2, j3, j4;
        //t = 0.0f; // ��� �������		
        int t = 0;
        float h = 1.0f;		
		for (int i = 0; i < size; i++) {	//16550
                v = (float) sqrt((float) pow(Vx, 2.0f) + (float) pow(Vy, 2.0f));
                //assert(Vx==0);
                if (Vx != 0) {
				//c 353 ?????????????????????????
                    w_ = w * L / v;//??????????????????????
                    betta_d = -(float) atan(Vy / Vx);//c 350
                } else {
                    w_ = w * L / v;
                    //betta_d = 0;
                    betta_d = -(float) atan(Vy / Vx);//c 350
                }
                
                Cxr = 0.01f * (1.0f + 170.0f * (T / L));// ��� ����� c 119
                Cyr = c1 * betta_d + c2 * w_ + c3 * betta_d * abs(betta_d);//c 323
                Cmr = m1 * betta_d + m2 * w_;

                Xr = Cxr * L * T * (float) pow(v, 2.0f) * p / 2.0f;//c 320
                Yr = Cyr * L * T * (float) pow(v, 2.0f) * p / 2.0f;
                Mr = Cmr * L * T * (float) pow(v, 2.0f) * p / 2.0f;

                K_betta = 0.43f * (float) pow(Ctt, -0.6f);
                fit = (float) pow(1.0f + Ctt, 0.508f);
                //IMPORTANT!!!  deltaR
                Yri = 3.14f * (deltaR - K_betta * xk * (betta_d + lr * w_)) * p * Ar * (float) pow(v * fik * fit, 2.0f) / (1.0f + 2.2f / (float) pow(lyamdaR, 2.0f / 3.0f));

                ld_ = 0.77f - 0.125f * (float) sqrt(Ctt) / (1.65f * (float) sqrt(Ctt) - 1.0f);
                betta_D = 1.22f - 0.0563f * (float) sqrt(Ctt) / (1.65f * (float) sqrt(Ctt) - 1.0f);
                fiD = 0.5f * ((float) sqrt(1.0f + 2.0f * Ctt / betta_D) + 1.0f);

                CyD1 = 12.0f * ld_ / (1.0f + 1.56f * ld_);
                CyD = CyD1 + 2.0f * betta_D * (float) pow(fiD, 2.0f);
                xD = xk * (CyD1 + 2.0f * betta_D * fiD) / (CyD1 + 2.0f * betta_D * (float) pow(fiD, 2.0f));
                A0 = 3.14f * (float) pow(D, 2.0f) / 4.0f;
                Ysi = CyD * (xD - 0.02f * xk) * (betta_d + lr * w_) * (p / 2.0f) * A0 * (float) pow(v, 2.0f) * (float) pow(fik, 2.0f);                
                ZpTei = 1000000.0f * (9.740f * (float) pow(No, 2.0f) - 2.23f * v); //�������//9.740f                
                ZpYri = 2.0f * (Yri - Ysi);//2 �����
               
                k1 = h * vxd(m,Vy, w, ZpTei, Xr, Kvx);
                q1 = h * vyd(m,Vx, w, ZpYri, Yr, Kvy);
                z1 = h * wd(lr,Mr, ZpTei, Yr, Kw);

                k2 = h * vxd(m,Vy + q1 / 2.0f, w + z1 / 2.0f, ZpTei, Xr, Kvx);
                q2 = h * vyd(m,Vx + k1 / 2.0f, w + z1 / 2.0f, ZpYri, Yr, Kvy);
                z2 = h * wd(lr,Mr, ZpYri, Yr, Kw);

                k3 = h * vxd(m,Vy + q2 / 2.0f, w + z2 / 2.0f, ZpTei, Xr, Kvx);
                q3 = h * vyd(m,Vx + k2 / 2.0f, w + z2 / 2.0f, ZpYri, Yr, Kvy);
                z3 = h * wd(lr,Mr, ZpYri, Yr, Kw);

                k4 = h * vxd(m,Vy + q3, w + z3, ZpTei, Xr, Kvx);
                q4 = h * vyd(m,Vx + k3, w + z3, ZpYri, Yr, Kvy);
                z4 = h * wd(lr,Mr, ZpYri, Yr, Kw);

                Vx = Vx + (1.0f / 6.0f) * (k1 + 2.0f * k2 + 2.0f * k3 + k4);
                //devVX[t] = Vx / 1.24f;
                devVX[t] = Vx;
                Vy = Vy + (1.0f / 6.0f) * (q1 + 2.0f * q2 + 2.0f * q3 + q4);
                devVY[t] = Vy;
                w = w + (1.0f / 6.0f) * (z1 + 2.0f * z2 + 2.0f * z3 + z4);
                devww[t] = w;

//---������ ��������-----------------------------------------
                k1 = h * vxd(m,Vy, w, ZpTei, Xr, Kvx);
                q1 = h * vyd(m,Vx, w, ZpYri, Yr, Kvy);
                z1 = h * wd(lr,Mr, ZpTei, Yr, Kw);

                k2 = h * vxd(m,Vy + q1 / 2.0f, w + z1 / 2.0f, ZpTei, Xr, Kvx);
                q2 = h * vyd(m,Vx + k1 / 2.0f, w + z1 / 2.0f, ZpYri, Yr, Kvy);
                z2 = h * wd(lr,Mr, ZpYri, Yr, Kw);

                k3 = h * vxd(m,Vy + q2 / 2.0f, w + z2 / 2.0f, ZpTei, Xr, Kvx);
                q3 = h * vyd(m,Vx + k2 / 2.0f, w + z2 / 2.0f, ZpYri, Yr, Kvy);
                z3 = h * wd(lr,Mr, ZpYri, Yr, Kw);

                k4 = h * vxd(m,Vy + q3, w + z3, ZpTei, Xr, Kvx);
                q4 = h * vyd(m,Vx + k3, w + z3, ZpYri, Yr, Kvy);
                z4 = h * wd(lr,Mr, ZpYri, Yr, Kw);

                devX[t] = Vx + (1.0f / 6.0f) * (k1 + 2.0f * k2 + 2.0f * k3 + k4);
                devY[t] = Vy + (1.0f / 6.0f) * (q1 + 2.0f * q2 + 2.0f * q3 + q4);
                devW[t] = w + (1.0f / 6.0f) * (z1 + 2.0f * z2 + 2.0f * z3 + z4);                
                //���� ��� ����������� ������� ���������                
                viv = devW[t];                                
                //-------
                Rmatr[0][0] = (float)cos(viv);
                Rmatr[0][1] = -(float)sin(viv);
                Rmatr[1][0] = (float)sin(viv);
                Rmatr[1][1] = (float)cos(viv);
                devXobs[t] = Rmatr[0][0] * devX[t] + Rmatr[0][1] * devY[t];
                devYobs[t] = Rmatr[1][0] * devX[t] + Rmatr[1][1] * devY[t];                
                //----------
                t++;               
                       
        }
		VX[j]=devVX[size-1];
		VY[j]=devVY[size-1];	
		ww[j]=devww[size-1];
		X[j]=devX[size-1];
		Y[j]=devY[size-1];
		W[j]=devW[size-1];
		Xobs[j]=devXobs[size-1];
		Yobs[j]=devYobs[size-1];		
	}
}