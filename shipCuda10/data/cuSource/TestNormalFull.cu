
#include <hip/hip_runtime.h>
extern "C"

__device__ void createNewNormal(int* x, int* y, int* z,int* nX, int* nY, int* nZ, int i)
{
int j = 0;
nX[i]=0;
nY[i]=0;
nZ[i]=0;
        for (int k = 0; k < 3; k++) {
            if (k < 2) {
                j = k + 1;
                nX[i] += -(y[k] - y[j]) * (z[k] + z[j]);
                nY[i] += -(z[k] - z[j]) * (x[k] + x[j]);
                nZ[i] += -(x[k] - x[j]) * (y[k] + y[j]);
            } else {
                j = 0;
                nX[i] += -(y[k] - y[j]) * (z[k] + z[j]);
                nY[i] += -(z[k] - z[j]) * (x[k] + x[j]);
                nZ[i] += -(x[k] - x[j]) * (y[k] + y[j]);
            }
        }
//return x;
}

extern "C"

__device__ void createNewNormalSecond(int* x, int* y, int* z,int* nX, int* nY, int* nZ, int i)
{
int j = 0;
int bufX=0,bufY=0,bufZ=0; 
        for (int k = 0; k < 3; k++) {
            if (k < 2) {
                j = k + 1;
                bufX += -(y[k] - y[j]) * (z[k] + z[j]);
                bufY += -(z[k] - z[j]) * (x[k] + x[j]);
                bufZ += -(x[k] - x[j]) * (y[k] + y[j]);
            } else {
                j = 0;
                bufX += -(y[k] - y[j]) * (z[k] + z[j]);
                bufY += -(z[k] - z[j]) * (x[k] + x[j]);
                bufZ += -(x[k] - x[j]) * (y[k] + y[j]);
            }
        }
nX[i]=bufX;
nY[i]=bufY;
nZ[i]=bufZ;
}


extern "C"
__global__ void TestNormalFull(int* point, int*polygon, int* nX, int* nY, int* nZ, int N) 

{ 
int i = blockDim.x * blockIdx.x + threadIdx.x; 

if (i < N){
int xxx[3];
int yyy[3];
int zzz[3];
for (int j=0;j<3;j++)
{
  xxx[j]= point[3*polygon[3*i+j]];
  yyy[j]= point[3*polygon[3*i+j]+1];
  zzz[j]= point[3*polygon[3*i+j]+2];
}

/*
xxx[0]= point[3*polygon[3*i]];
xxx[1]= point[3*polygon[3*i+1]];
xxx[2]= point[3*polygon[3*i+2]];
yyy[0]= point[3*polygon[3*i]+1];
yyy[1]= point[3*polygon[3*i+1]+1];
yyy[2]= point[3*polygon[3*i+2]+1];
zzz[0]= point[3*polygon[3*i]+2];
zzz[1]= point[3*polygon[3*i+1]+2];
zzz[2]= point[3*polygon[3*i+2]+2];
*/

createNewNormal(xxx,yyy,zzz,nX,nY,nZ,i);
}
}